#include "hip/hip_runtime.h"
//
// # Yocto/CuTrace: Path tracing on Cuda/Optix
//
// Yocto/CuTrace is a simple path tracer written on the Yocto/Scene model.
// Yocto/CuTrace is implemented in `yocto_cutrace.h`, `yocto_cutrace.cpp`,
// and `yocto_cutrace.cu`.
// This library includes a stand-alone implementaton of the PCG32 random number
// generator by M.E. O'Neill.
//
// THIS IS AN EXPERIMENTAL LIBRARY THAT IS NOT READY FOR PRIME TIME
//

//
// LICENSE:
//
// Copyright (c) 2016 -- 2022 Fabio Pellacini
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
//
//
// LICENSE OF INCLUDED SOFTWARE for Pcg random number generator
//
// This code also includes a small exerpt from http://www.pcg-random.org/
// licensed as follows
// *Really* minimal PCG32 code / (c) 2014 M.E. O'Neill / pcg-random.org
// Licensed under Apache License 2.0 (NO WARRANTY, etc. see website)
//

#include <optix_device.h>

#include "yocto_color.h"
#include "yocto_geometry.h"
#include "yocto_math.h"
#include "yocto_sampling.h"
#include "yocto_shading.h"

// HACK TO ALLOW CUT&PASTING FROM YOCTO'S CODE
#define inline __forceinline__ __device__
#define static static __forceinline__ __device__
#define optix_shader extern "C" __global__
#define optix_constant extern "C" __constant__

// whether to use builtin compound types or yocto's ones
#define CUTRACE_BUILTIN_VECS 0

// -----------------------------------------------------------------------------
// SUBSTITUTES FOR STD TYPES
// -----------------------------------------------------------------------------
namespace yocto {

// pair
template <typename T1, typename T2>
struct pair_ {
  T1 first;
  T2 second;
};

template <typename T>
struct span {
  inline bool   empty() const { return _size == 0; }
  inline size_t size() const { return _size; }

  inline T&       operator[](int idx) { return _data[idx]; }
  inline const T& operator[](int idx) const { return _data[idx]; }
  inline T&       at(int idx) { return _data[idx]; }
  inline const T& at(int idx) const { return _data[idx]; }

  inline T*       begin() { return _data; }
  inline T*       end() { return _data + _size; }
  inline const T* begin() const { return _data; }
  inline const T* end() const { return _data + _size; }

  inline T&       front() { return *_data; }
  inline T&       back() { return *(_data + _size - 1); }
  inline const T& front() const { return *_data; }
  inline const T& back() const { return *(_data + _size - 1); }

  inline T*       data() { return _data; }
  inline const T* data() const { return _data; }

  T*     _data = nullptr;
  size_t _size = 0;
};

}  // namespace yocto

// -----------------------------------------------------------------------------
// SAMPLING FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// simplified version of possible implementation from cpprenference.com
template <class T>
static const T* _upper_bound(const T* first, const T* last, const T& value) {
  const T*  it;
  ptrdiff_t count, step;
  count = last - first;

  while (count > 0) {
    it   = first;
    step = count / 2;
    it += step;
    if (!(value < *it)) {
      first = ++it;
      count -= step + 1;
    } else
      count = step;
  }
  return first;
}

// Sample a discrete distribution represented by its cdf.
inline int sample_discrete(const span<float>& cdf, float r) {
  r = clamp(r * cdf.back(), (float)0, cdf.back() - (float)0.00001);
  auto idx =
      (int)(_upper_bound(cdf.data(), cdf.data() + cdf.size(), r) - cdf.data());
  return clamp(idx, 0, (int)cdf.size() - 1);
}
// Pdf for uniform discrete distribution sampling.
inline float sample_discrete_pdf(const span<float>& cdf, int idx) {
  if (idx == 0) return cdf.at(0);
  return cdf.at(idx) - cdf.at(idx - 1);
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// CUDA HELPERS
// -----------------------------------------------------------------------------
namespace yocto {

template <typename T>
struct cuspan {
  inline bool     empty() const { return _size == 0; }
  inline size_t   size() const { return _size; }
  inline T&       operator[](int idx) { return _data[idx]; }
  inline const T& operator[](int idx) const { return _data[idx]; }

  inline T*       begin() { return _data; }
  inline T*       end() { return _data + _size; }
  inline const T* begin() const { return _data; }
  inline const T* end() const { return _data + _size; }

  inline T&       front() { return *_data; }
  inline T&       back() { return *(_data + _size - 1); }
  inline const T& front() const { return *_data; }
  inline const T& back() const { return *(_data + _size - 1); }

  inline operator span<T>() const { return {_data, _size}; }

  T*     _data = nullptr;
  size_t _size = 0;
};

template <typename T, size_t Size = 16>
struct svector {
  inline bool     empty() const { return _size == 0; }
  inline size_t   size() const { return _size; }
  inline T&       operator[](int idx) { return _data[idx]; }
  inline const T& operator[](int idx) const { return _data[idx]; }

  inline T*       begin() { return _data; }
  inline T*       end() { return _data + _size; }
  inline const T* begin() const { return _data; }
  inline const T* end() const { return _data + _size; }

  inline T&       front() { return *_data; }
  inline T&       back() { return *(_data + _size - 1); }
  inline const T& front() const { return *_data; }
  inline const T& back() const { return *(_data + _size - 1); }

  inline void push_back(const T& value) { _data[_size++] = value; }
  inline void pop_back() { _size--; }

  T      _data[Size] = {};
  size_t _size       = 0;
};

inline void* unpackPointer(uint32_t i0, uint32_t i1) {
  const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
  void*          ptr  = reinterpret_cast<void*>(uptr);
  return ptr;
}

inline void packPointer(void* ptr, uint32_t& i0, uint32_t& i1) {
  const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
  i0                  = uptr >> 32;
  i1                  = uptr & 0x00000000ffffffff;
}

template <typename T>
inline T* getPRD() {
  const uint32_t u0 = optixGetPayload_0();
  const uint32_t u1 = optixGetPayload_1();
  return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// CUTRACE TYPES
// -----------------------------------------------------------------------------
namespace yocto {

constexpr int invalidid = -1;

struct cutrace_state {
  int               width            = 0;
  int               height           = 0;
  int               samples          = 0;
  cuspan<vec4f>     image            = {};
  cuspan<vec3f>     albedo           = {};
  cuspan<vec3f>     normal           = {};
  cuspan<int>       hits             = {};
  cuspan<rng_state> rngs             = {};
  cuspan<vec4f>     denoised         = {};
  cuspan<byte>      denoiser_state   = {};
  cuspan<byte>      denoiser_scratch = {};
};

struct cucamera_data {
  frame3f frame        = {};
  float   lens         = {};
  float   film         = {};
  float   aspect       = {};
  float   focus        = {};
  float   aperture     = {};
  bool    orthographic = {};
};

struct cutexture_data {
  int                 width   = 0;
  int                 height  = 0;
  bool                linear  = false;
  hipTextureObject_t texture = 0;
  hipArray_t         array   = nullptr;
};

enum struct material_type {
  // clang-format off
  matte, glossy, reflective, transparent, refractive, subsurface, volumetric, 
  gltfpbr
  // clang-format on
};

struct cumaterial_data {
  material_type type         = material_type::matte;
  vec3f         emission     = {0, 0, 0};
  vec3f         color        = {0, 0, 0};
  float         roughness    = 0;
  float         metallic     = 0;
  float         ior          = 1.5f;
  vec3f         scattering   = {0, 0, 0};
  float         scanisotropy = 0;
  float         trdepth      = 0.01f;
  float         opacity      = 1;

  int emission_tex   = invalidid;
  int color_tex      = invalidid;
  int roughness_tex  = invalidid;
  int scattering_tex = invalidid;
  int normal_tex     = invalidid;
};

struct cuinstance_data {
  frame3f frame    = {{1, 0, 0}, {0, 1, 0}, {0, 0, 1}, {0, 0, 0}};
  int     shape    = -1;
  int     material = -1;
};

struct cushape_data {
  cuspan<vec3f> positions = {};
  cuspan<vec3f> normals   = {};
  cuspan<vec2f> texcoords = {};
  cuspan<vec4f> colors    = {};
  cuspan<vec3i> triangles = {};
};

struct cuenvironment_data {
  frame3f frame        = {{1, 0, 0}, {0, 1, 0}, {0, 0, 1}, {0, 0, 0}};
  vec3f   emission     = {0, 0, 0};
  int     emission_tex = invalidid;
};

struct cuscene_data {
  cuspan<cucamera_data>      cameras      = {};
  cuspan<cutexture_data>     textures     = {};
  cuspan<cumaterial_data>    materials    = {};
  cuspan<cushape_data>       shapes       = {};
  cuspan<cuinstance_data>    instances    = {};
  cuspan<cuenvironment_data> environments = {};
};

// Type of tracing algorithm
enum struct trace_sampler_type {
  path,        // path tracing
  pathdirect,  // path tracing with direct
  pathmis,     // path tracing with mis
  pathtest,    // path tracing for testing
  naive,       // naive path tracing
  eyelight,    // eyelight rendering
  furnace,     // furnace test
  falsecolor,  // false color rendering
};
// Type of false color visualization
enum struct trace_falsecolor_type {
  // clang-format off
  position, normal, frontfacing, gnormal, gfrontfacing, texcoord, mtype, color,
  emission, roughness, opacity, metallic, delta, instance, shape, material, 
  element, highlight
  // clang-format on
};

// Default trace seed
constexpr auto trace_default_seed = 961748941ull;

// params
struct trace_params {
  int                   camera         = 0;
  int                   resolution     = 1280;
  trace_sampler_type    sampler        = trace_sampler_type::path;
  trace_falsecolor_type falsecolor     = trace_falsecolor_type::color;
  int                   samples        = 512;
  int                   bounces        = 8;
  float                 clamp          = 10;
  bool                  nocaustics     = false;
  bool                  envhidden      = false;
  bool                  tentfilter     = false;
  uint64_t              seed           = trace_default_seed;
  bool                  embreebvh      = false;
  bool                  highqualitybvh = false;
  bool                  noparallel     = false;
  int                   pratio         = 8;
  bool                  denoise        = false;
  int                   batch          = 1;
};

using cutrace_bvh = OptixTraversableHandle;

// light
struct cutrace_light {
  int           instance     = invalidid;
  int           environment  = invalidid;
  cuspan<float> elements_cdf = {};
};

// lights
struct cutrace_lights {
  cuspan<cutrace_light> lights = {};
};

struct cutrace_globals {
  cutrace_state          state  = {};
  cuscene_data           scene  = {};
  OptixTraversableHandle bvh    = 0;
  cutrace_lights         lights = {};
  trace_params           params = {};
};

// global data
optix_constant cutrace_globals globals;

// compatibility aliases
using trace_bvh    = cutrace_bvh;
using trace_lights = cutrace_lights;

}  // namespace yocto

// -----------------------------------------------------------------------------
// SCENE FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// compatibility aliases
using scene_data       = cuscene_data;
using camera_data      = cucamera_data;
using material_data    = cumaterial_data;
using texture_data     = cutexture_data;
using instance_data    = cuinstance_data;
using shape_data       = cushape_data;
using environment_data = cuenvironment_data;

// constant values
constexpr auto min_roughness = 0.03f * 0.03f;

// Evaluates an image at a point `uv`.
static vec4f eval_texture(const texture_data& texture, const vec2f& texcoord,
    bool as_linear = false, bool no_interpolation = false,
    bool clamp_to_edge = false) {
  auto fromTexture = tex2D<float4>(texture.texture, texcoord.x, texcoord.y);
  auto color       = vec4f{
      fromTexture.x, fromTexture.y, fromTexture.z, fromTexture.w};
  if (as_linear && !texture.linear) {
    return srgb_to_rgb(color);
  } else {
    return color;
  }
}

// Helpers
static vec4f eval_texture(const scene_data& scene, int texture, const vec2f& uv,
    bool ldr_as_linear = false, bool no_interpolation = false,
    bool clamp_to_edge = false) {
  if (texture == invalidid) return {1, 1, 1, 1};
  return eval_texture(
      scene.textures[texture], uv, ldr_as_linear, no_interpolation);
}

// Material parameters evaluated at a point on the surface
struct material_point {
  material_type type         = material_type::gltfpbr;
  vec3f         emission     = {0, 0, 0};
  vec3f         color        = {0, 0, 0};
  float         opacity      = 1;
  float         roughness    = 0;
  float         metallic     = 0;
  float         ior          = 1;
  vec3f         density      = {0, 0, 0};
  vec3f         scattering   = {0, 0, 0};
  float         scanisotropy = 0;
  float         trdepth      = 0.01f;
};

// Eval position
static vec3f eval_position(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return transform_point(
        instance.frame, interpolate_triangle(shape.positions[t.x],
                            shape.positions[t.y], shape.positions[t.z], uv));
  } else {
    return {0, 0, 0};
  }
}

// Shape element normal.
static vec3f eval_element_normal(
    const scene_data& scene, const instance_data& instance, int element) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return transform_normal(
        instance.frame, triangle_normal(shape.positions[t.x],
                            shape.positions[t.y], shape.positions[t.z]));
  } else {
    return {0, 0, 0};
  }
}

// Eval normal
static vec3f eval_normal(const scene_data& scene, const instance_data& instance,
    int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (shape.normals.empty())
    return eval_element_normal(scene, instance, element);
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return transform_normal(
        instance.frame, normalize(interpolate_triangle(shape.normals[t.x],
                            shape.normals[t.y], shape.normals[t.z], uv)));
  } else {
    return {0, 0, 0};
  }
}

// Eval texcoord
static vec2f eval_texcoord(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (shape.texcoords.empty()) return uv;
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return interpolate_triangle(
        shape.texcoords[t.x], shape.texcoords[t.y], shape.texcoords[t.z], uv);
  } else {
    return {0, 0};
  }
}

// Shape element normal.
static pair_<vec3f, vec3f> eval_element_tangents(
    const scene_data& scene, const instance_data& instance, int element) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty() && !shape.texcoords.empty()) {
    auto t   = shape.triangles[element];
    auto tuv = triangle_tangents_fromuv(shape.positions[t.x],
        shape.positions[t.y], shape.positions[t.z], shape.texcoords[t.x],
        shape.texcoords[t.y], shape.texcoords[t.z]);
    return {transform_direction(instance.frame, tuv.first),
        transform_direction(instance.frame, tuv.second)};
  } else {
    return {};
  }
}

static vec3f eval_normalmap(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& shape    = scene.shapes[instance.shape];
  auto& material = scene.materials[instance.material];
  // apply normal mapping
  auto normal   = eval_normal(scene, instance, element, uv);
  auto texcoord = eval_texcoord(scene, instance, element, uv);
  if (material.normal_tex != invalidid && (!shape.triangles.empty())) {
    auto& normal_tex = scene.textures[material.normal_tex];
    auto  normalmap  = -1 + 2 * xyz(eval_texture(normal_tex, texcoord, false));
    auto  tuv        = eval_element_tangents(scene, instance, element);
    auto  frame      = frame3f{tuv.first, tuv.second, normal, {0, 0, 0}};
    frame.x          = orthonormalize(frame.x, frame.z);
    frame.y          = normalize(cross(frame.z, frame.x));
    auto flip_v      = dot(frame.y, tuv.second) < 0;
    normalmap.y *= flip_v ? 1 : -1;  // flip vertical axis
    normal = transform_normal(frame, normalmap);
  }
  return normal;
}

// Eval shading position
static vec3f eval_shading_position(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv,
    const vec3f& outgoing) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty()) {
    return eval_position(scene, instance, element, uv);
  } else {
    return {0, 0, 0};
  }
}

// Eval shading normal
static vec3f eval_shading_normal(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv,
    const vec3f& outgoing) {
  auto& shape    = scene.shapes[instance.shape];
  auto& material = scene.materials[instance.material];
  if (!shape.triangles.empty()) {
    auto normal = eval_normal(scene, instance, element, uv);
    if (material.normal_tex != invalidid) {
      normal = eval_normalmap(scene, instance, element, uv);
    }
    if (material.type == material_type::refractive) return normal;
    return dot(normal, outgoing) >= 0 ? normal : -normal;
  } else {
    return {0, 0, 0};
  }
}

// Eval color
static vec4f eval_color(const scene_data& scene, const instance_data& instance,
    int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (shape.colors.empty()) return {1, 1, 1, 1};
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return interpolate_triangle(
        shape.colors[t.x], shape.colors[t.y], shape.colors[t.z], uv);
  } else {
    return {0, 0, 0, 0};
  }
}

// Evaluate material
static material_point eval_material(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& material = scene.materials[instance.material];
  auto  texcoord = eval_texcoord(scene, instance, element, uv);

  // evaluate textures
  auto emission_tex = eval_texture(
      scene, material.emission_tex, texcoord, true);
  auto color_shp     = eval_color(scene, instance, element, uv);
  auto color_tex     = eval_texture(scene, material.color_tex, texcoord, true);
  auto roughness_tex = eval_texture(
      scene, material.roughness_tex, texcoord, false);
  auto scattering_tex = eval_texture(
      scene, material.scattering_tex, texcoord, true);

  // material point
  auto point         = material_point{};
  point.type         = material.type;
  point.emission     = material.emission * xyz(emission_tex);
  point.color        = material.color * xyz(color_tex) * xyz(color_shp);
  point.opacity      = material.opacity * color_tex.w * color_shp.w;
  point.metallic     = material.metallic * roughness_tex.z;
  point.roughness    = material.roughness * roughness_tex.y;
  point.roughness    = point.roughness * point.roughness;
  point.ior          = material.ior;
  point.scattering   = material.scattering * xyz(scattering_tex);
  point.scanisotropy = material.scanisotropy;
  point.trdepth      = material.trdepth;

  // volume density
  if (material.type == material_type::refractive ||
      material.type == material_type::volumetric ||
      material.type == material_type::subsurface) {
    point.density = -log(clamp(point.color, 0.0001f, 1.0f)) / point.trdepth;
  } else {
    point.density = {0, 0, 0};
  }

  // fix roughness
  if (point.type == material_type::matte ||
      point.type == material_type::gltfpbr ||
      point.type == material_type::glossy) {
    point.roughness = clamp(point.roughness, min_roughness, 1.0f);
  } else if (material.type == material_type::volumetric) {
    point.roughness = 0;
  } else {
    if (point.roughness < min_roughness) point.roughness = 0;
  }

  return point;
}

static bool is_volumetric(const material_data& material) {
  return material.type == material_type::refractive ||
         material.type == material_type::volumetric ||
         material.type == material_type::subsurface;
}

// check if an instance is volumetric
static bool is_volumetric(
    const scene_data& scene, const instance_data& instance) {
  return is_volumetric(scene.materials[instance.material]);
}

// check if a brdf is a delta
static bool is_delta(const material_point& material) {
  return (material.type == material_type::reflective &&
             material.roughness == 0) ||
         (material.type == material_type::refractive &&
             material.roughness == 0) ||
         (material.type == material_type::transparent &&
             material.roughness == 0) ||
         (material.type == material_type::volumetric);
}

static ray3f eval_camera(
    const cucamera_data& camera, const vec2f& image_uv, const vec2f& lens_uv) {
  auto film = camera.aspect >= 1
                  ? vec2f{camera.film, camera.film / camera.aspect}
                  : vec2f{camera.film * camera.aspect, camera.film};
  auto q    = vec3f{
      film.x * (0.5f - image_uv.x), film.y * (image_uv.y - 0.5f), camera.lens};
  // ray direction through the lens center
  auto dc = -normalize(q);
  // point on the lens
  auto e = vec3f{
      lens_uv.x * camera.aperture / 2, lens_uv.y * camera.aperture / 2, 0};
  // point on the focus plane
  auto p = dc * camera.focus / abs(dc.z);
  // correct ray direction to account for camera focusing
  auto d = normalize(p - e);
  // done
  return ray3f{
      transform_point(camera.frame, e), transform_direction(camera.frame, d)};
}

// Evaluate environment color.
static vec3f eval_environment(const scene_data& scene,
    const environment_data& environment, const vec3f& direction) {
  auto wl       = transform_direction_inverse(environment.frame, direction);
  auto texcoord = vec2f{
      atan2(wl.z, wl.x) / (2 * pif), acos(clamp(wl.y, -1.0f, 1.0f)) / pif};
  if (texcoord.x < 0) texcoord.x += 1;
  return environment.emission *
         xyz(eval_texture(scene, environment.emission_tex, texcoord));
}

// Evaluate all environment color.
static vec3f eval_environment(const scene_data& scene, const vec3f& direction) {
  auto emission = vec3f{0, 0, 0};
  for (auto& environment : scene.environments) {
    emission += eval_environment(scene, environment, direction);
  }
  return emission;
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// RAY-SCENE INTERSECTION
// -----------------------------------------------------------------------------
namespace yocto {

// intersection result
struct scene_intersection {
  int   instance = -1;
  int   element  = -1;
  vec2f uv       = {0, 0};
  float distance = 0;
  bool  hit      = false;
  float _pad     = 0;
};

// closest hit
optix_shader void __closesthit__intersect_scene() {
  auto& intersection    = *getPRD<scene_intersection>();
  intersection.instance = optixGetInstanceIndex();
  intersection.element  = optixGetPrimitiveIndex();
  intersection.uv       = {
      optixGetTriangleBarycentrics().x, optixGetTriangleBarycentrics().y};
  intersection.distance = optixGetRayTmax();
  intersection.hit      = true;
}

// anyhit shader
optix_shader void __anyhit__intersect_scene() {}

// miss shader
optix_shader void __miss__intersect_scene() {
  auto& intersection    = *getPRD<scene_intersection>();
  intersection.instance = 0;
  intersection.element  = 0;
  intersection.uv       = {0, 0};
  intersection.distance = optixGetRayTmax();
  intersection.hit      = false;
}

// scene intersection via shaders
static scene_intersection intersect_scene(
    const trace_bvh& bvh, const cuscene_data& scene, const ray3f& ray) {
  auto     intersection = scene_intersection{};
  uint32_t u0, u1;
  packPointer(&intersection, u0, u1);
  optixTrace(bvh, {ray.o.x, ray.o.y, ray.o.z}, {ray.d.x, ray.d.y, ray.d.z},
      ray.tmin, ray.tmax, 0.0f, OptixVisibilityMask(255),
      OPTIX_RAY_FLAG_DISABLE_ANYHIT, 0, 0, 0, u0, u1);
  return intersection;
}

// instance intersection, for now manual
static scene_intersection intersect_instance(const trace_bvh& bvh,
    const cuscene_data& scene, int instance_id, const ray3f& ray) {
  auto& instance     = scene.instances[instance_id];
  auto& shape        = scene.shapes[instance.shape];
  auto  intersection = scene_intersection{};
  auto  tray         = ray3f{transform_point_inverse(instance.frame, ray.o),
      transform_vector_inverse(instance.frame, ray.d)};
  for (auto element = 0; element < shape.triangles.size(); element++) {
    auto& triangle = shape.triangles[element];
    auto  isec     = intersect_triangle(tray, shape.positions[triangle.x],
             shape.positions[triangle.y], shape.positions[triangle.z]);
    if (!isec.hit) continue;
    intersection.hit      = true;
    intersection.instance = instance_id;
    intersection.element  = element;
    intersection.uv       = isec.uv;
    intersection.distance = isec.distance;
    tray.tmax             = isec.distance;
  }
  return intersection;
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// TRACE FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// Convenience functions
[[maybe_unused]] static vec3f eval_position(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_position(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static vec3f eval_normal(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_normal(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static vec3f eval_element_normal(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_element_normal(
      scene, scene.instances[intersection.instance], intersection.element);
}
[[maybe_unused]] static vec3f eval_shading_position(const scene_data& scene,
    const scene_intersection& intersection, const vec3f& outgoing) {
  return eval_shading_position(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv, outgoing);
}
[[maybe_unused]] static vec3f eval_shading_normal(const scene_data& scene,
    const scene_intersection& intersection, const vec3f& outgoing) {
  return eval_shading_normal(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv, outgoing);
}
[[maybe_unused]] static vec2f eval_texcoord(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_texcoord(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static material_point eval_material(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_material(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static bool is_volumetric(
    const scene_data& scene, const scene_intersection& intersection) {
  return is_volumetric(scene, scene.instances[intersection.instance]);
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// TRACE FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// Evaluates/sample the BRDF scaled by the cosine of the incoming direction.
static vec3f eval_emission(const material_point& material, const vec3f& normal,
    const vec3f& outgoing) {
  return dot(normal, outgoing) >= 0 ? material.emission : vec3f{0, 0, 0};
}

// Evaluates/sample the BRDF scaled by the cosine of the incoming direction.
static vec3f eval_bsdfcos(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness == 0) return {0, 0, 0};

  if (material.type == material_type::matte) {
    return eval_matte(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::glossy) {
    return eval_glossy(material.color, material.ior, material.roughness, normal,
        outgoing, incoming);
  } else if (material.type == material_type::reflective) {
    return eval_reflective(
        material.color, material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return eval_transparent(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return eval_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::subsurface) {
    return eval_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::gltfpbr) {
    return eval_gltfpbr(material.color, material.ior, material.roughness,
        material.metallic, normal, outgoing, incoming);
  } else {
    return {0, 0, 0};
  }
}

static vec3f eval_delta(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness != 0) return {0, 0, 0};

  if (material.type == material_type::reflective) {
    return eval_reflective(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return eval_transparent(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return eval_refractive(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::volumetric) {
    return eval_passthrough(material.color, normal, outgoing, incoming);
  } else {
    return {0, 0, 0};
  }
}

// Picks a direction based on the BRDF
static vec3f sample_bsdfcos(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, float rnl, const vec2f& rn) {
  if (material.roughness == 0) return {0, 0, 0};

  if (material.type == material_type::matte) {
    return sample_matte(material.color, normal, outgoing, rn);
  } else if (material.type == material_type::glossy) {
    return sample_glossy(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::reflective) {
    return sample_reflective(
        material.color, material.roughness, normal, outgoing, rn);
  } else if (material.type == material_type::transparent) {
    return sample_transparent(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::refractive) {
    return sample_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::subsurface) {
    return sample_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::gltfpbr) {
    return sample_gltfpbr(material.color, material.ior, material.roughness,
        material.metallic, normal, outgoing, rnl, rn);
  } else {
    return {0, 0, 0};
  }
}

static vec3f sample_delta(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, float rnl) {
  if (material.roughness != 0) return {0, 0, 0};

  if (material.type == material_type::reflective) {
    return sample_reflective(material.color, normal, outgoing);
  } else if (material.type == material_type::transparent) {
    return sample_transparent(
        material.color, material.ior, normal, outgoing, rnl);
  } else if (material.type == material_type::refractive) {
    return sample_refractive(
        material.color, material.ior, normal, outgoing, rnl);
  } else if (material.type == material_type::volumetric) {
    return sample_passthrough(material.color, normal, outgoing);
  } else {
    return {0, 0, 0};
  }
}

// Compute the weight for sampling the BRDF
static float sample_bsdfcos_pdf(const material_point& material,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness == 0) return 0;

  if (material.type == material_type::matte) {
    return sample_matte_pdf(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::glossy) {
    return sample_glossy_pdf(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::reflective) {
    return sample_reflective_pdf(
        material.color, material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return sample_tranparent_pdf(material.color, material.ior,
        material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return sample_refractive_pdf(material.color, material.ior,
        material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::subsurface) {
    return sample_refractive_pdf(material.color, material.ior,
        material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::gltfpbr) {
    return sample_gltfpbr_pdf(material.color, material.ior, material.roughness,
        material.metallic, normal, outgoing, incoming);
  } else {
    return 0;
  }
}

static float sample_delta_pdf(const material_point& material,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness != 0) return 0;

  if (material.type == material_type::reflective) {
    return sample_reflective_pdf(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return sample_tranparent_pdf(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return sample_refractive_pdf(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::volumetric) {
    return sample_passthrough_pdf(material.color, normal, outgoing, incoming);
  } else {
    return 0;
  }
}

static vec3f eval_scattering(const material_point& material,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.density == vec3f{0, 0, 0}) return {0, 0, 0};
  return material.scattering * material.density *
         eval_phasefunction(material.scanisotropy, outgoing, incoming);
}

static vec3f sample_scattering(const material_point& material,
    const vec3f& outgoing, float rnl, const vec2f& rn) {
  if (material.density == vec3f{0, 0, 0}) return {0, 0, 0};
  return sample_phasefunction(material.scanisotropy, outgoing, rn);
}

static float sample_scattering_pdf(const material_point& material,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.density == vec3f{0, 0, 0}) return 0;
  return sample_phasefunction_pdf(material.scanisotropy, outgoing, incoming);
}

// Sample camera
static ray3f sample_camera(const camera_data& camera, const vec2i& ij,
    const vec2i& image_size, const vec2f& puv, const vec2f& luv, bool tent) {
  if (!tent) {
    auto uv = vec2f{
        (ij.x + puv.x) / image_size.x, (ij.y + puv.y) / image_size.y};
    return eval_camera(camera, uv, sample_disk(luv));
  } else {
    const auto width  = 2.0f;
    const auto offset = 0.5f;
    auto       fuv =
        width *
            vec2f{
                puv.x < 0.5f ? sqrt(2 * puv.x) - 1 : 1 - sqrt(2 - 2 * puv.x),
                puv.y < 0.5f ? sqrt(2 * puv.y) - 1 : 1 - sqrt(2 - 2 * puv.y),
            } +
        offset;
    auto uv = vec2f{
        (ij.x + fuv.x) / image_size.x, (ij.y + fuv.y) / image_size.y};
    return eval_camera(camera, uv, sample_disk(luv));
  }
}

// Sample lights wrt solid angle
static vec3f sample_lights(const scene_data& scene, const trace_lights& lights,
    const vec3f& position, float rl, float rel, const vec2f& ruv) {
  auto  light_id = sample_uniform((int)lights.lights.size(), rl);
  auto& light    = lights.lights[light_id];
  if (light.instance != invalidid) {
    auto& instance  = scene.instances[light.instance];
    auto& shape     = scene.shapes[instance.shape];
    auto  element   = sample_discrete(light.elements_cdf, rel);
    auto  uv        = (!shape.triangles.empty()) ? sample_triangle(ruv) : ruv;
    auto  lposition = eval_position(scene, instance, element, uv);
    return normalize(lposition - position);
  } else if (light.environment != invalidid) {
    auto& environment = scene.environments[light.environment];
    if (environment.emission_tex != invalidid) {
      auto& emission_tex = scene.textures[environment.emission_tex];
      auto  idx          = sample_discrete(light.elements_cdf, rel);
      auto  uv = vec2f{((idx % emission_tex.width) + 0.5f) / emission_tex.width,
          ((idx / emission_tex.width) + 0.5f) / emission_tex.height};
      return transform_direction(environment.frame,
          {cos(uv.x * 2 * pif) * sin(uv.y * pif), cos(uv.y * pif),
              sin(uv.x * 2 * pif) * sin(uv.y * pif)});
    } else {
      return sample_sphere(ruv);
    }
  } else {
    return {0, 0, 0};
  }
}

// Sample lights pdf
static float sample_lights_pdf(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const vec3f& position, const vec3f& direction) {
  auto pdf = 0.0f;
  for (auto& light : lights.lights) {
    if (light.instance != invalidid) {
      auto& instance = scene.instances[light.instance];
      // check all intersection
      auto lpdf          = 0.0f;
      auto next_position = position;
      for (auto bounce = 0; bounce < 100; bounce++) {
        auto intersection = intersect_instance(
            bvh, scene, light.instance, {next_position, direction});
        if (!intersection.hit) break;
        // accumulate pdf
        auto lposition = eval_position(
            scene, instance, intersection.element, intersection.uv);
        auto lnormal = eval_element_normal(
            scene, instance, intersection.element);
        // prob triangle * area triangle = area triangle mesh
        auto area = light.elements_cdf.back();
        lpdf += distance_squared(lposition, position) /
                (abs(dot(lnormal, direction)) * area);
        // continue
        next_position = lposition + direction * 1e-3f;
      }
      pdf += lpdf;
    } else if (light.environment != invalidid) {
      auto& environment = scene.environments[light.environment];
      if (environment.emission_tex != invalidid) {
        auto& emission_tex = scene.textures[environment.emission_tex];
        auto  wl = transform_direction_inverse(environment.frame, direction);
        auto  texcoord = vec2f{atan2(wl.z, wl.x) / (2 * pif),
            acos(clamp(wl.y, -1.0f, 1.0f)) / pif};
        if (texcoord.x < 0) texcoord.x += 1;
        auto i = clamp(
            (int)(texcoord.x * emission_tex.width), 0, emission_tex.width - 1);
        auto j    = clamp((int)(texcoord.y * emission_tex.height), 0,
               emission_tex.height - 1);
        auto prob = sample_discrete_pdf(
                        light.elements_cdf, j * emission_tex.width + i) /
                    light.elements_cdf.back();
        auto angle = (2 * pif / emission_tex.width) *
                     (pif / emission_tex.height) *
                     sin(pif * (j + 0.5f) / emission_tex.height);
        pdf += prob / angle;
      } else {
        pdf += 1 / (4 * pif);
      }
    }
  }
  pdf *= sample_uniform_pdf((int)lights.lights.size());
  return pdf;
}

struct trace_result {
  vec3f radiance = {0, 0, 0};
  bool  hit      = false;
  vec3f albedo   = {0, 0, 0};
  vec3f normal   = {0, 0, 0};
};

// Recursive path tracing.
static trace_result trace_path(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const ray3f& ray_, rng_state& rng,
    const trace_params& params) {
  // initialize
  auto radiance      = vec3f{0, 0, 0};
  auto weight        = vec3f{1, 1, 1};
  auto ray           = ray_;
  auto volume_stack  = svector<material_point>{};
  auto max_roughness = 0.0f;
  auto hit           = false;
  auto hit_albedo    = vec3f{0, 0, 0};
  auto hit_normal    = vec3f{0, 0, 0};
  auto opbounce      = 0;

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // handle transmission if inside a volume
    auto in_volume = false;
    if (!volume_stack.empty()) {
      auto& vsdf     = volume_stack.back();
      auto  distance = sample_transmittance(
           vsdf.density, intersection.distance, rand1f(rng), rand1f(rng));
      weight *= eval_transmittance(vsdf.density, distance) /
                sample_transmittance_pdf(
                    vsdf.density, distance, intersection.distance);
      in_volume             = distance < intersection.distance;
      intersection.distance = distance;
    }

    // switch between surface and volume
    if (!in_volume) {
      // prepare shading point
      auto outgoing = -ray.d;
      auto position = eval_shading_position(scene, intersection, outgoing);
      auto normal   = eval_shading_normal(scene, intersection, outgoing);
      auto material = eval_material(scene, intersection);

      // correct roughness
      if (params.nocaustics) {
        max_roughness      = max(material.roughness, max_roughness);
        material.roughness = max_roughness;
      }

      // handle opacity
      if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
        if (opbounce++ > 128) break;
        ray = {position + ray.d * 1e-2f, ray.d};
        bounce -= 1;
        continue;
      }

      // set hit variables
      if (bounce == 0) {
        hit        = true;
        hit_albedo = material.color;
        hit_normal = normal;
      }

      // accumulate emission
      radiance += weight * eval_emission(material, normal, outgoing);

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (!is_delta(material)) {
        if (rand1f(rng) < 0.5f) {
          incoming = sample_bsdfcos(
              material, normal, outgoing, rand1f(rng), rand2f(rng));
        } else {
          incoming = sample_lights(
              scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
        }
        if (incoming == vec3f{0, 0, 0}) break;
        weight *=
            eval_bsdfcos(material, normal, outgoing, incoming) /
            (0.5f * sample_bsdfcos_pdf(material, normal, outgoing, incoming) +
                0.5f *
                    sample_lights_pdf(scene, bvh, lights, position, incoming));
      } else {
        incoming = sample_delta(material, normal, outgoing, rand1f(rng));
        weight *= eval_delta(material, normal, outgoing, incoming) /
                  sample_delta_pdf(material, normal, outgoing, incoming);
      }

      // update volume stack
      if (is_volumetric(scene, intersection) &&
          dot(normal, outgoing) * dot(normal, incoming) < 0) {
        if (volume_stack.empty()) {
          auto material = eval_material(scene, intersection);
          volume_stack.push_back(material);
        } else {
          volume_stack.pop_back();
        }
      }

      // setup next iteration
      ray = {position, incoming};
    } else {
      // prepare shading point
      auto  outgoing = -ray.d;
      auto  position = ray.o + ray.d * intersection.distance;
      auto& vsdf     = volume_stack.back();

      // accumulate emission
      // radiance += weight * eval_volemission(emission, outgoing);

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (rand1f(rng) < 0.5f) {
        incoming = sample_scattering(vsdf, outgoing, rand1f(rng), rand2f(rng));
      } else {
        incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
      }
      if (incoming == vec3f{0, 0, 0}) break;
      weight *=
          eval_scattering(vsdf, outgoing, incoming) /
          (0.5f * sample_scattering_pdf(vsdf, outgoing, incoming) +
              0.5f * sample_lights_pdf(scene, bvh, lights, position, incoming));

      // setup next iteration
      ray = {position, incoming};
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Recursive path tracing.
static trace_result trace_pathdirect(const scene_data& scene,
    const trace_bvh& bvh, const trace_lights& lights, const ray3f& ray_,
    rng_state& rng, const trace_params& params) {
  // initialize
  auto radiance      = vec3f{0, 0, 0};
  auto weight        = vec3f{1, 1, 1};
  auto ray           = ray_;
  auto volume_stack  = svector<material_point>{};
  auto max_roughness = 0.0f;
  auto hit           = false;
  auto hit_albedo    = vec3f{0, 0, 0};
  auto hit_normal    = vec3f{0, 0, 0};
  auto next_emission = true;
  auto opbounce      = 0;

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if ((bounce > 0 || !params.envhidden) && next_emission)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // handle transmission if inside a volume
    auto in_volume = false;
    if (!volume_stack.empty()) {
      auto& vsdf     = volume_stack.back();
      auto  distance = sample_transmittance(
           vsdf.density, intersection.distance, rand1f(rng), rand1f(rng));
      weight *= eval_transmittance(vsdf.density, distance) /
                sample_transmittance_pdf(
                    vsdf.density, distance, intersection.distance);
      in_volume             = distance < intersection.distance;
      intersection.distance = distance;
    }

    // switch between surface and volume
    if (!in_volume) {
      // prepare shading point
      auto outgoing = -ray.d;
      auto position = eval_shading_position(scene, intersection, outgoing);
      auto normal   = eval_shading_normal(scene, intersection, outgoing);
      auto material = eval_material(scene, intersection);

      // correct roughness
      if (params.nocaustics) {
        max_roughness      = max(material.roughness, max_roughness);
        material.roughness = max_roughness;
      }

      // handle opacity
      if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
        if (opbounce++ > 128) break;
        ray = {position + ray.d * 1e-2f, ray.d};
        bounce -= 1;
        continue;
      }

      // set hit variables
      if (bounce == 0) {
        hit        = true;
        hit_albedo = material.color;
        hit_normal = normal;
      }

      // accumulate emission
      if (next_emission)
        radiance += weight * eval_emission(material, normal, outgoing);

      // direct
      if (!is_delta(material)) {
        auto incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
        auto pdf = sample_lights_pdf(scene, bvh, lights, position, incoming);
        auto bsdfcos = eval_bsdfcos(material, normal, outgoing, incoming);
        if (bsdfcos != vec3f{0, 0, 0} && pdf > 0) {
          auto intersection = intersect_scene(bvh, scene, {position, incoming});
          auto emission =
              !intersection.hit
                  ? eval_environment(scene, incoming)
                  : eval_emission(eval_material(scene,
                                      scene.instances[intersection.instance],
                                      intersection.element, intersection.uv),
                        eval_shading_normal(scene,
                            scene.instances[intersection.instance],
                            intersection.element, intersection.uv, -incoming),
                        -incoming);
          radiance += weight * bsdfcos * emission / pdf;
        }
        next_emission = false;
      } else {
        next_emission = true;
      }

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (!is_delta(material)) {
        if (rand1f(rng) < 0.5f) {
          incoming = sample_bsdfcos(
              material, normal, outgoing, rand1f(rng), rand2f(rng));
        } else {
          incoming = sample_lights(
              scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
        }
        if (incoming == vec3f{0, 0, 0}) break;
        weight *=
            eval_bsdfcos(material, normal, outgoing, incoming) /
            (0.5f * sample_bsdfcos_pdf(material, normal, outgoing, incoming) +
                0.5f *
                    sample_lights_pdf(scene, bvh, lights, position, incoming));
      } else {
        incoming = sample_delta(material, normal, outgoing, rand1f(rng));
        if (incoming == vec3f{0, 0, 0}) break;
        weight *= eval_delta(material, normal, outgoing, incoming) /
                  sample_delta_pdf(material, normal, outgoing, incoming);
      }

      // update volume stack
      if (is_volumetric(scene, intersection) &&
          dot(normal, outgoing) * dot(normal, incoming) < 0) {
        if (volume_stack.empty()) {
          auto material = eval_material(scene, intersection);
          volume_stack.push_back(material);
        } else {
          volume_stack.pop_back();
        }
      }

      // setup next iteration
      ray = {position, incoming};
    } else {
      // prepare shading point
      auto  outgoing = -ray.d;
      auto  position = ray.o + ray.d * intersection.distance;
      auto& vsdf     = volume_stack.back();

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (rand1f(rng) < 0.5f) {
        incoming = sample_scattering(vsdf, outgoing, rand1f(rng), rand2f(rng));
      } else {
        incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
      }
      if (incoming == vec3f{0, 0, 0}) break;
      weight *=
          eval_scattering(vsdf, outgoing, incoming) /
          (0.5f * sample_scattering_pdf(vsdf, outgoing, incoming) +
              0.5f * sample_lights_pdf(scene, bvh, lights, position, incoming));

      // setup next iteration
      ray = {position, incoming};
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Recursive path tracing with MIS.
static trace_result trace_pathmis(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const ray3f& ray_, rng_state& rng,
    const trace_params& params) {
  // initialize
  auto radiance      = vec3f{0, 0, 0};
  auto weight        = vec3f{1, 1, 1};
  auto ray           = ray_;
  auto volume_stack  = svector<material_point>{};
  auto max_roughness = 0.0f;
  auto hit           = false;
  auto hit_albedo    = vec3f{0, 0, 0};
  auto hit_normal    = vec3f{0, 0, 0};
  auto opbounce      = 0;

  // MIS helpers
  auto mis_heuristic = [](float this_pdf, float other_pdf) {
    return (this_pdf * this_pdf) /
           (this_pdf * this_pdf + other_pdf * other_pdf);
  };
  auto next_emission     = true;
  auto next_intersection = scene_intersection{};

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // intersect next point
    auto intersection = next_emission ? intersect_scene(bvh, scene, ray)
                                      : next_intersection;
    if (!intersection.hit) {
      if ((bounce > 0 || !params.envhidden) && next_emission)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // handle transmission if inside a volume
    auto in_volume = false;
    if (!volume_stack.empty()) {
      auto& vsdf     = volume_stack.back();
      auto  distance = sample_transmittance(
           vsdf.density, intersection.distance, rand1f(rng), rand1f(rng));
      weight *= eval_transmittance(vsdf.density, distance) /
                sample_transmittance_pdf(
                    vsdf.density, distance, intersection.distance);
      in_volume             = distance < intersection.distance;
      intersection.distance = distance;
    }

    // switch between surface and volume
    if (!in_volume) {
      // prepare shading point
      auto outgoing = -ray.d;
      auto position = eval_shading_position(scene, intersection, outgoing);
      auto normal   = eval_shading_normal(scene, intersection, outgoing);
      auto material = eval_material(scene, intersection);

      // correct roughness
      if (params.nocaustics) {
        max_roughness      = max(material.roughness, max_roughness);
        material.roughness = max_roughness;
      }

      // handle opacity
      if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
        if (opbounce++ > 128) break;
        ray = {position + ray.d * 1e-2f, ray.d};
        bounce -= 1;
        continue;
      }

      // set hit variables
      if (bounce == 0) {
        hit        = true;
        hit_albedo = material.color;
        hit_normal = normal;
      }

      // accumulate emission
      if (next_emission) {
        radiance += weight * eval_emission(material, normal, outgoing);
      }

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (!is_delta(material)) {
        // direct with MIS --- light
        for (auto sample_light : {true, false}) {
          incoming = sample_light ? sample_lights(scene, lights, position,
                                        rand1f(rng), rand1f(rng), rand2f(rng))
                                  : sample_bsdfcos(material, normal, outgoing,
                                        rand1f(rng), rand2f(rng));
          if (incoming == vec3f{0, 0, 0}) break;
          auto bsdfcos   = eval_bsdfcos(material, normal, outgoing, incoming);
          auto light_pdf = sample_lights_pdf(
              scene, bvh, lights, position, incoming);
          auto bsdf_pdf = sample_bsdfcos_pdf(
              material, normal, outgoing, incoming);
          auto mis_weight = sample_light
                                ? mis_heuristic(light_pdf, bsdf_pdf) / light_pdf
                                : mis_heuristic(bsdf_pdf, light_pdf) / bsdf_pdf;
          if (bsdfcos != vec3f{0, 0, 0} && mis_weight != 0) {
            auto intersection = intersect_scene(
                bvh, scene, {position, incoming});
            if (!sample_light) next_intersection = intersection;
            auto emission = vec3f{0, 0, 0};
            if (!intersection.hit) {
              emission = eval_environment(scene, incoming);
            } else {
              auto material = eval_material(scene,
                  scene.instances[intersection.instance], intersection.element,
                  intersection.uv);
              emission      = eval_emission(material,
                       eval_shading_normal(scene,
                           scene.instances[intersection.instance],
                           intersection.element, intersection.uv, -incoming),
                       -incoming);
            }
            radiance += weight * bsdfcos * emission * mis_weight;
          }
        }

        // indirect
        weight *= eval_bsdfcos(material, normal, outgoing, incoming) /
                  sample_bsdfcos_pdf(material, normal, outgoing, incoming);
        next_emission = false;
      } else {
        incoming = sample_delta(material, normal, outgoing, rand1f(rng));
        weight *= eval_delta(material, normal, outgoing, incoming) /
                  sample_delta_pdf(material, normal, outgoing, incoming);
        next_emission = true;
      }

      // update volume stack
      if (is_volumetric(scene, intersection) &&
          dot(normal, outgoing) * dot(normal, incoming) < 0) {
        if (volume_stack.empty()) {
          auto material = eval_material(scene, intersection);
          volume_stack.push_back(material);
        } else {
          volume_stack.pop_back();
        }
      }

      // setup next iteration
      ray = {position, incoming};
    } else {
      // prepare shading point
      auto  outgoing = -ray.d;
      auto  position = ray.o + ray.d * intersection.distance;
      auto& vsdf     = volume_stack.back();

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (rand1f(rng) < 0.5f) {
        incoming = sample_scattering(vsdf, outgoing, rand1f(rng), rand2f(rng));
        next_emission = true;
      } else {
        incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
        next_emission = true;
      }
      weight *=
          eval_scattering(vsdf, outgoing, incoming) /
          (0.5f * sample_scattering_pdf(vsdf, outgoing, incoming) +
              0.5f * sample_lights_pdf(scene, bvh, lights, position, incoming));

      // setup next iteration
      ray = {position, incoming};
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Recursive path tracing.
static trace_result trace_pathtest(const scene_data& scene,
    const trace_bvh& bvh, const trace_lights& lights, const ray3f& ray_,
    rng_state& rng, const trace_params& params) {
  // initialize
  auto radiance      = vec3f{0, 0, 0};
  auto weight        = vec3f{1, 1, 1};
  auto ray           = ray_;
  auto max_roughness = 0.0f;
  auto hit           = false;
  auto hit_albedo    = vec3f{0, 0, 0};
  auto hit_normal    = vec3f{0, 0, 0};
  auto opbounce      = 0;

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // prepare shading point
    auto outgoing = -ray.d;
    auto position = eval_shading_position(scene, intersection, outgoing);
    auto normal   = eval_shading_normal(scene, intersection, outgoing);
    auto material = eval_material(scene, intersection);
    material.type = material_type::matte;

    // set hit variables
    if (bounce == 0) {
      hit        = true;
      hit_albedo = material.color;
      hit_normal = normal;
    }

    // accumulate emission
    radiance += weight * eval_emission(material, normal, outgoing);

    // next direction
    auto incoming = vec3f{0, 0, 0};
    if (!is_delta(material)) {
      if (rand1f(rng) < 0.5f) {
        incoming = sample_bsdfcos(
            material, normal, outgoing, rand1f(rng), rand2f(rng));
      } else {
        incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
      }
      if (incoming == vec3f{0, 0, 0}) break;
      weight *=
          eval_bsdfcos(material, normal, outgoing, incoming) /
          (0.5f * sample_bsdfcos_pdf(material, normal, outgoing, incoming) +
              0.5f * sample_lights_pdf(scene, bvh, lights, position, incoming));
    } else {
      incoming = sample_delta(material, normal, outgoing, rand1f(rng));
      weight *= eval_delta(material, normal, outgoing, incoming) /
                sample_delta_pdf(material, normal, outgoing, incoming);
    }

    // setup next iteration
    ray = {position, incoming};

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Recursive path tracing.
static trace_result trace_naive(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const ray3f& ray_, rng_state& rng,
    const trace_params& params) {
  // initialize
  auto radiance   = vec3f{0, 0, 0};
  auto weight     = vec3f{1, 1, 1};
  auto ray        = ray_;
  auto hit        = false;
  auto hit_albedo = vec3f{0, 0, 0};
  auto hit_normal = vec3f{0, 0, 0};
  auto opbounce   = 0;

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // prepare shading point
    auto outgoing = -ray.d;
    auto position = eval_shading_position(scene, intersection, outgoing);
    auto normal   = eval_shading_normal(scene, intersection, outgoing);
    auto material = eval_material(scene, intersection);

    // handle opacity
    if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
      if (opbounce++ > 128) break;
      ray = {position + ray.d * 1e-2f, ray.d};
      bounce -= 1;
      continue;
    }

    // set hit variables
    if (bounce == 0) {
      hit        = true;
      hit_albedo = material.color;
      hit_normal = normal;
    }

    // accumulate emission
    radiance += weight * eval_emission(material, normal, outgoing);

    // next direction
    auto incoming = vec3f{0, 0, 0};
    if (material.roughness != 0) {
      incoming = sample_bsdfcos(
          material, normal, outgoing, rand1f(rng), rand2f(rng));
      if (incoming == vec3f{0, 0, 0}) break;
      weight *= eval_bsdfcos(material, normal, outgoing, incoming) /
                sample_bsdfcos_pdf(material, normal, outgoing, incoming);
    } else {
      incoming = sample_delta(material, normal, outgoing, rand1f(rng));
      if (incoming == vec3f{0, 0, 0}) break;
      weight *= eval_delta(material, normal, outgoing, incoming) /
                sample_delta_pdf(material, normal, outgoing, incoming);
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }

    // setup next iteration
    ray = {position, incoming};
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Eyelight for quick previewing.
static trace_result trace_eyelight(const scene_data& scene,
    const trace_bvh& bvh, const trace_lights& lights, const ray3f& ray_,
    rng_state& rng, const trace_params& params) {
  // initialize
  auto radiance   = vec3f{0, 0, 0};
  auto weight     = vec3f{1, 1, 1};
  auto ray        = ray_;
  auto hit        = false;
  auto hit_albedo = vec3f{0, 0, 0};
  auto hit_normal = vec3f{0, 0, 0};
  auto opbounce   = 0;

  // trace  path
  for (auto bounce = 0; bounce < max(params.bounces, 4); bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // prepare shading point
    auto outgoing = -ray.d;
    auto position = eval_shading_position(scene, intersection, outgoing);
    auto normal   = eval_shading_normal(scene, intersection, outgoing);
    auto material = eval_material(scene, intersection);

    // handle opacity
    if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
      if (opbounce++ > 128) break;
      ray = {position + ray.d * 1e-2f, ray.d};
      bounce -= 1;
      continue;
    }

    // set hit variables
    if (bounce == 0) {
      hit        = true;
      hit_albedo = material.color;
      hit_normal = normal;
    }

    // accumulate emission
    auto incoming = outgoing;
    radiance += weight * eval_emission(material, normal, outgoing);

    // brdf * light
    radiance += weight * pif *
                eval_bsdfcos(material, normal, outgoing, incoming);

    // continue path
    if (!is_delta(material)) break;
    incoming = sample_delta(material, normal, outgoing, rand1f(rng));
    if (incoming == vec3f{0, 0, 0}) break;
    weight *= eval_delta(material, normal, outgoing, incoming) /
              sample_delta_pdf(material, normal, outgoing, incoming);
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // setup next iteration
    ray = {position, incoming};
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Furnace test.
static trace_result trace_furnace(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const ray3f& ray_, rng_state& rng,
    const trace_params& params) {
  // initialize
  auto radiance   = vec3f{0, 0, 0};
  auto weight     = vec3f{1, 1, 1};
  auto ray        = ray_;
  auto hit        = false;
  auto hit_albedo = vec3f{0, 0, 0};
  auto hit_normal = vec3f{0, 0, 0};
  auto opbounce   = 0;
  auto in_volume  = false;

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // exit loop
    if (bounce > 0 && !in_volume) {
      radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // prepare shading point
    auto  outgoing = -ray.d;
    auto& instance = scene.instances[intersection.instance];
    auto  element  = intersection.element;
    auto  uv       = intersection.uv;
    auto  position = eval_position(scene, instance, element, uv);
    auto  normal = eval_shading_normal(scene, instance, element, uv, outgoing);
    auto  material = eval_material(scene, instance, element, uv);

    // handle opacity
    if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
      if (opbounce++ > 128) break;
      ray = {position + ray.d * 1e-2f, ray.d};
      bounce -= 1;
      continue;
    }

    // set hit variables
    if (bounce == 0) {
      hit        = true;
      hit_albedo = material.color;
      hit_normal = normal;
    }

    // accumulate emission
    radiance += weight * eval_emission(material, normal, outgoing);

    // next direction
    auto incoming = vec3f{0, 0, 0};
    if (material.roughness != 0) {
      incoming = sample_bsdfcos(
          material, normal, outgoing, rand1f(rng), rand2f(rng));
      if (incoming == vec3f{0, 0, 0}) break;
      weight *= eval_bsdfcos(material, normal, outgoing, incoming) /
                sample_bsdfcos_pdf(material, normal, outgoing, incoming);
    } else {
      incoming = sample_delta(material, normal, outgoing, rand1f(rng));
      if (incoming == vec3f{0, 0, 0}) break;
      weight *= eval_delta(material, normal, outgoing, incoming) /
                sample_delta_pdf(material, normal, outgoing, incoming);
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }

    // update volume stack
    if (dot(normal, outgoing) * dot(normal, incoming) < 0)
      in_volume = !in_volume;

    // setup next iteration
    ray = {position, incoming};
  }

  // done
  return {radiance, hit, hit_albedo, hit_normal};
}

// False color rendering
static trace_result trace_falsecolor(const scene_data& scene,
    const trace_bvh& bvh, const trace_lights& lights, const ray3f& ray,
    rng_state& rng, const trace_params& params) {
  // intersect next point
  auto intersection = intersect_scene(bvh, scene, ray);
  if (!intersection.hit) return {};

  // prepare shading point
  auto outgoing = -ray.d;
  auto position = eval_shading_position(scene, intersection, outgoing);
  auto normal   = eval_shading_normal(scene, intersection, outgoing);
  auto gnormal  = eval_element_normal(scene, intersection);
  auto texcoord = eval_texcoord(scene, intersection);
  auto material = eval_material(scene, intersection);
  auto delta    = is_delta(material) ? 1.0f : 0.0f;

  // hash color
  auto hashed_color = [](int id) {
    auto rng = make_rng(trace_default_seed, id * 2 + 1);
    return pow(0.5f + 0.5f * rand3f(rng), 2.2f);
  };

  // compute result
  auto result = vec3f{0, 0, 0};
  switch (params.falsecolor) {
    case trace_falsecolor_type::position:
      result = position * 0.5f + 0.5f;
      break;
    case trace_falsecolor_type::normal: result = normal * 0.5f + 0.5f; break;
    case trace_falsecolor_type::frontfacing:
      result = dot(normal, -ray.d) > 0 ? vec3f{0, 1, 0} : vec3f{1, 0, 0};
      break;
    case trace_falsecolor_type::gnormal: result = gnormal * 0.5f + 0.5f; break;
    case trace_falsecolor_type::gfrontfacing:
      result = dot(gnormal, -ray.d) > 0 ? vec3f{0, 1, 0} : vec3f{1, 0, 0};
      break;
    case trace_falsecolor_type::mtype:
      result = hashed_color((int)material.type);
      break;
    case trace_falsecolor_type::texcoord:
      result = {fmod(texcoord.x, 1.0f), fmod(texcoord.y, 1.0f), 0};
      break;
    case trace_falsecolor_type::color: result = material.color; break;
    case trace_falsecolor_type::emission: result = material.emission; break;
    case trace_falsecolor_type::roughness:
      result = {material.roughness, material.roughness, material.roughness};
      break;
    case trace_falsecolor_type::opacity:
      result = {material.opacity, material.opacity, material.opacity};
      break;
    case trace_falsecolor_type::metallic:
      result = {material.metallic, material.metallic, material.metallic};
      break;
    case trace_falsecolor_type::delta: result = {delta, delta, delta}; break;
    case trace_falsecolor_type::element:
      result = hashed_color(intersection.element);
      break;
    case trace_falsecolor_type::instance:
      result = hashed_color(intersection.instance);
      break;
    case trace_falsecolor_type::shape:
      result = hashed_color(scene.instances[intersection.instance].shape);
      break;
    case trace_falsecolor_type::material:
      result = hashed_color(scene.instances[intersection.instance].material);
      break;
    case trace_falsecolor_type::highlight: {
      if (material.emission == vec3f{0, 0, 0})
        material.emission = {0.2f, 0.2f, 0.2f};
      result = material.emission * abs(dot(-ray.d, normal));
      break;
    } break;
    default: result = {0, 0, 0};
  }

  // done
  return {srgb_to_rgb(result), true, material.color, normal};
}

static trace_result trace_sampler(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const ray3f& ray, rng_state& rng,
    const trace_params& params) {
  switch (params.sampler) {
    case trace_sampler_type::path:
      return trace_path(scene, bvh, lights, ray, rng, params);
    case trace_sampler_type::pathdirect:
      return trace_pathdirect(scene, bvh, lights, ray, rng, params);
    case trace_sampler_type::pathmis:
      return trace_pathmis(scene, bvh, lights, ray, rng, params);
    case trace_sampler_type::pathtest:
      return trace_pathtest(scene, bvh, lights, ray, rng, params);
    case trace_sampler_type::naive:
      return trace_naive(scene, bvh, lights, ray, rng, params);
    case trace_sampler_type::eyelight:
      return trace_eyelight(scene, bvh, lights, ray, rng, params);
    case trace_sampler_type::furnace:
      return trace_furnace(scene, bvh, lights, ray, rng, params);
    case trace_sampler_type::falsecolor:
      return trace_falsecolor(scene, bvh, lights, ray, rng, params);
    default: {
      return {};
    }
  }
}

static void trace_sample(cutrace_state& state, const cuscene_data& scene,
    const cutrace_bvh& bvh, const cutrace_lights& lights, int i, int j,
    int sample, const trace_params& params) {
  auto& camera = scene.cameras[params.camera];
  // auto  sampler = get_trace_sampler_func(params);
  auto idx    = state.width * j + i;
  auto ray    = sample_camera(camera, {i, j}, {state.width, state.height},
         rand2f(state.rngs[idx]), rand2f(state.rngs[idx]), params.tentfilter);
  auto result = trace_sampler(scene, bvh, lights, ray, state.rngs[idx], params);
  // auto [radiance, hit, albedo, normal] = sampler(
  //    scene, bvh, lights, ray, state.rngs[idx], params);
  auto radiance = result.radiance;
  auto hit      = result.hit;
  auto albedo   = result.albedo;
  auto normal   = result.normal;
  if (!isfinite(radiance)) radiance = {0, 0, 0};
  if (max(radiance) > params.clamp)
    radiance = radiance * (params.clamp / max(radiance));
  auto weight = 1.0f / (sample + 1);
  if (hit) {
    state.image[idx] = lerp(
        state.image[idx], {radiance.x, radiance.y, radiance.z, 1}, weight);
    state.albedo[idx] = lerp(state.albedo[idx], albedo, weight);
    state.normal[idx] = lerp(state.normal[idx], normal, weight);
    state.hits[idx] += 1;
  } else if (!params.envhidden && !scene.environments.empty()) {
    state.image[idx] = lerp(
        state.image[idx], {radiance.x, radiance.y, radiance.z, 1}, weight);
    state.albedo[idx] = lerp(state.albedo[idx], {1, 1, 1}, weight);
    state.normal[idx] = lerp(state.normal[idx], -ray.d, weight);
    state.hits[idx] += 1;
  } else {
    state.image[idx]  = lerp(state.image[idx], {0, 0, 0, 0}, weight);
    state.albedo[idx] = lerp(state.albedo[idx], {0, 0, 0}, weight);
    state.normal[idx] = lerp(state.normal[idx], -ray.d, weight);
  }
}

// raygen shader
optix_shader void __raygen__trace_pixel() {
  // pixel index
  auto ij  = optixGetLaunchIndex();
  auto idx = ij.y * globals.state.width + ij.x;

  // initialize state on first sample
  if (globals.state.samples == 0) {
    globals.state.image[idx] = {0, 0, 0, 0};
    globals.state.rngs[idx]  = make_rng(98273987, idx * 2 + 1);
  }

  // run shading
  auto ssample  = globals.state.samples;
  auto nsamples = globals.params.batch;
  for (auto sample = ssample; sample < ssample + nsamples; sample++) {
    trace_sample(globals.state, globals.scene, globals.bvh, globals.lights,
        optixGetLaunchIndex().x, optixGetLaunchIndex().y, sample,
        globals.params);
  }
}

}  // namespace yocto